#include "hip/hip_runtime.h"
#include "separableConvolution.h"
#include "hip/hip_runtime.h"
#include "book.h"
#include <math.h>

#define KERNEL_RADIUS 15
#define KERNEL_LENGTH (2 * KERNEL_RADIUS + 1)

__constant__ float c_Kernel[ KERNEL_LENGTH ];

void setConvolutionKernel_31( float *h_Kernel )
{
    hipMemcpyToSymbol(HIP_SYMBOL( c_Kernel), h_Kernel, KERNEL_LENGTH * sizeof(float) );
}

// how many threads per block in x (total num threads: x*y)
#define	ROWS_BLOCKDIM_X 16
// how many threads per block in y
#define	ROWS_BLOCKDIM_Y 4
// how many pixels in x are convolved by each thread
#define	ROWS_RESULT_STEPS 8
// these are the border pixels (loaded to support the kernel width for processing)
// the effective border width is ROWS_HALO_STEPS * ROWS_BLOCKDIM_X, which has to be
// larger or equal to the kernel radius to work
#define	ROWS_HALO_STEPS 1

#define	COLUMNS_BLOCKDIM_X 16
#define	COLUMNS_BLOCKDIM_Y 16
#define	COLUMNS_RESULT_STEPS 8
#define	COLUMNS_HALO_STEPS 1

#define	DEPTH_BLOCKDIM_X 16
#define	DEPTH_BLOCKDIM_Z 16
#define	DEPTH_RESULT_STEPS 8
#define	DEPTH_HALO_STEPS 1

extern "C" int multipleOfX_31()
{
	return imax( DEPTH_BLOCKDIM_X, imax( ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X, COLUMNS_BLOCKDIM_X) );
}
extern "C" int multipleOfY_31()
{
	return imax( ROWS_BLOCKDIM_Y, COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y );
}
extern "C" int multipleOfZ_31()
{
	return DEPTH_RESULT_STEPS * DEPTH_BLOCKDIM_Z;
}

extern "C" int convolve_31( float *image, float *kernelX, float *kernelY, float *kernelZ, int imageW, int imageH, int imageD, int convolveX, int convolveY, int convolveZ, int devCUDA )
{
	fprintf(stderr, "Cuda device: %i\n", devCUDA );

	// test dimensions
	if ( imageW % multipleOfX_31() != 0 ||
		 imageH % multipleOfY_31() != 0 ||
		 imageD % multipleOfZ_31() != 0 )
		return 0; //false

	fprintf(stderr, "Convolving X: %i\n", convolveX );
	fprintf(stderr, "Convolving Y: %i\n", convolveY );
	fprintf(stderr, "Convolving Z: %i\n", convolveZ );

	fprintf(stderr, "Image Size X: %i\n", imageW );
	fprintf(stderr, "Image Size Y: %i\n", imageH );
	fprintf(stderr, "Image Size Z: %i\n", imageD );

	float *d_Input, *d_Output;

	hipSetDevice( devCUDA );

	// allocate memory for CUDA
	HANDLE_ERROR( hipMalloc((void **)&d_Input,   imageW * imageH * imageD * sizeof(float)) );
	HANDLE_ERROR( hipMalloc((void **)&d_Output,  imageW * imageH * imageD * sizeof(float)) );

    // copy input to graphics card
	HANDLE_ERROR( hipMemcpy(d_Input, image, imageW * imageH * imageD * sizeof(float), hipMemcpyHostToDevice) );

	int in = 0;

    if ( convolveX != 0 )
    {
        HANDLE_ERROR( hipDeviceSynchronize() );
		setConvolutionKernel_31( kernelX );
	    HANDLE_ERROR( hipDeviceSynchronize() );
		convolutionX_31( d_Output, d_Input, imageW, imageH, imageD );
		in = 1;
    }

    if ( convolveY != 0 )
    {
        HANDLE_ERROR( hipDeviceSynchronize() );
    	setConvolutionKernel_31( kernelY );
        HANDLE_ERROR( hipDeviceSynchronize() );

    	if ( in == 0 )
    	{
    		convolutionY_31( d_Output, d_Input, imageW, imageH, imageD );
    		in = 1;
    	}
    	else
    	{
    		convolutionY_31( d_Input, d_Output, imageW, imageH, imageD );
    		in = 0;
    	}
    }

    if ( convolveZ != 0 )
    {
        HANDLE_ERROR( hipDeviceSynchronize() );
		setConvolutionKernel_31( kernelZ );
	    HANDLE_ERROR( hipDeviceSynchronize() );

		if ( in == 0 )
		{
			convolutionZ_31( d_Output, d_Input, imageW, imageH, imageD );
			in = 1;
		}
		else
		{
			convolutionZ_31( d_Input, d_Output, imageW, imageH, imageD );
			in = 0;
		}
    }

    HANDLE_ERROR( hipDeviceSynchronize() );

    // copy back
    if ( in == 1 )
    	HANDLE_ERROR( hipMemcpy(image, d_Output, imageW * imageH * imageD * sizeof(float), hipMemcpyDeviceToHost) );
    else
    	HANDLE_ERROR( hipMemcpy(image, d_Input, imageW * imageH * imageD * sizeof(float), hipMemcpyDeviceToHost) );

    HANDLE_ERROR( hipFree(d_Output) );
    HANDLE_ERROR( hipFree(d_Input) );

    hipDeviceReset();

    return -1; // true
}

__global__ void convolutionX_31_Kernel( float *d_Dst, float *d_Src, int imageW, int imageH, int imageD )
{
    __shared__ float s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

    const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;
    const int baseZ = blockIdx.z;

    // set the input and output arrays to the right offset (actually the output is not at the right offset, but this is corrected later)
    d_Src += baseZ * imageH * imageW + baseY * imageW + baseX;
    d_Dst += baseZ * imageH * imageW + baseY * imageW + baseX;

    // Load main data
    // Start copying after the ROWS_HALO_STEPS, only the original data that will be convolved
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = d_Src[i * ROWS_BLOCKDIM_X];
    }

    // Load left halo
    // If the data fetched is outside of the image (note: baseX can be <0 for the first block) , use a zero-out of bounds strategy
#pragma unroll

    for (int i = 0; i < ROWS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
    }

    //Load right halo
#pragma unroll

    for (int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
    }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        float sum = 0;

#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j];
        }

        d_Dst[i * ROWS_BLOCKDIM_X] = sum;
    }
}

__global__ void convolutionY_31_Kernel( float *d_Dst, float *d_Src, int imageW, int imageH, int imageD )
{
    __shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

    //Offset to the upper halo edge
    const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
    const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
    const int baseZ = blockIdx.z;

    d_Src += baseZ * imageH * imageW + baseY * imageW + baseX;
    d_Dst += baseZ * imageH * imageW + baseY * imageW + baseX;

    //Main data
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = d_Src[i * COLUMNS_BLOCKDIM_Y * imageW];
    }

    //Upper halo
#pragma unroll

    for (int i = 0; i < COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * imageW] : 0;
    }

    //Lower halo
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y]= (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * imageW] : 0;
    }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
        float sum = 0;
#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];
        }

        d_Dst[i * COLUMNS_BLOCKDIM_Y * imageW] = sum;
    }
}

__global__ void convolutionZ_31_Kernel( float *d_Dst, float *d_Src, int imageW, int imageH, int imageD )
{
	// here it is [x][z], we leave out y as it has a size of 1
    __shared__ float s_Data[DEPTH_BLOCKDIM_X][(DEPTH_RESULT_STEPS + 2 * DEPTH_HALO_STEPS) * DEPTH_BLOCKDIM_Z + 1];

    //Offset to the upper halo edge
    const int baseX = blockIdx.x * DEPTH_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y;
    const int baseZ = (blockIdx.z * DEPTH_RESULT_STEPS - DEPTH_HALO_STEPS) * DEPTH_BLOCKDIM_Z + threadIdx.z;

    d_Src += baseZ * imageH * imageW + baseY * imageW + baseX;
    d_Dst += baseZ * imageH * imageW + baseY * imageW + baseX;

    //Main data
#pragma unroll

    for (int i = DEPTH_HALO_STEPS; i < DEPTH_HALO_STEPS + DEPTH_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z] = d_Src[i * DEPTH_BLOCKDIM_Z * imageW * imageH];
    }

    //Upper halo
#pragma unroll

    for (int i = 0; i < DEPTH_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z] = (baseZ >= -i * DEPTH_BLOCKDIM_Z) ? d_Src[i * DEPTH_BLOCKDIM_Z * imageW * imageH] : 0;
    }

    //Lower halo
#pragma unroll

    for (int i = DEPTH_HALO_STEPS + DEPTH_RESULT_STEPS; i < DEPTH_HALO_STEPS + DEPTH_RESULT_STEPS + DEPTH_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z]= (imageD - baseZ > i * DEPTH_BLOCKDIM_Z) ? d_Src[i * DEPTH_BLOCKDIM_Z * imageW * imageH] : 0;
    }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = DEPTH_HALO_STEPS; i < DEPTH_HALO_STEPS + DEPTH_RESULT_STEPS; i++)
    {
        float sum = 0;
#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.x][threadIdx.z + i * DEPTH_BLOCKDIM_Z + j];
        }

        d_Dst[i * DEPTH_BLOCKDIM_Z * imageW * imageH] = sum;
    }
}

void convolutionX_31( float *d_Dst, float *d_Src, int imageW, int imageH, int imageD )
{
    dim3 blocks(imageW / (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X), imageH / ROWS_BLOCKDIM_Y, imageD);
    dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y, 1);

    convolutionX_31_Kernel<<<blocks, threads>>>( d_Dst, d_Src, imageW, imageH, imageD );
}

void convolutionY_31( float *d_Dst, float *d_Src, int imageW, int imageH, int imageD )
{
    dim3 blocks(imageW / COLUMNS_BLOCKDIM_X, imageH / (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y), imageD);
    dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y, 1);

    convolutionY_31_Kernel<<<blocks, threads>>>( d_Dst, d_Src, imageW, imageH, imageD );
}

void convolutionZ_31( float *d_Dst, float *d_Src, int imageW, int imageH, int imageD )
{
    dim3 blocks(imageW / DEPTH_BLOCKDIM_X, imageH, imageD/ (DEPTH_RESULT_STEPS * DEPTH_BLOCKDIM_Z) );
    dim3 threads(DEPTH_BLOCKDIM_X, 1, DEPTH_BLOCKDIM_Z);

    convolutionZ_31_Kernel<<<blocks, threads>>>( d_Dst, d_Src, imageW, imageH, imageD );
}
